
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <random>
#include <chrono>
#include <thread>
#include <vector>

using namespace std::chrono;

// Initialize host array x of length dim with random floats in [0,1)
void random_init(float* x, int dim) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (int i = 0; i < dim; ++i) 
        x[i] = dist(gen);
}

// GPU AXPY kernel: c[i] = alpha * a[i] + b[i]
__global__
void axpyKernel(const float* a, const float* b, float* c, float alpha, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Stride loop for cases where grid*block < N
    while (i < N) {
        c[i] = alpha * a[i] + b[i];
        i += gridDim.x * blockDim.x;
    }
}

// Multithreaded CPU version of AXPY using std::thread
void axpy_cpu(const float* a, const float* b, float* c, float alpha, int N) {
    unsigned int num_threads = std::thread::hardware_concurrency();
    if (num_threads == 0) num_threads = 4;  // Fallback if detection fails

    int chunk = (N + num_threads - 1) / num_threads;
    std::vector<std::thread> threads;
    threads.reserve(num_threads);

    // Launch threads, each handling a contiguous chunk
    for (unsigned int t = 0; t < num_threads; ++t) {
        int start = t * chunk;
        int end   = std::min(start + chunk, N);
        threads.emplace_back([=]() {
            for (int i = start; i < end; ++i) {
                c[i] = alpha * a[i] + b[i];
            }
        });
    }
    // Wait for all to finish
    for (auto& th : threads) {
        th.join();
    }
}

int main() {
    // Open results file (append mode)
    std::ofstream results("REZULTS.txt", std::ios::out | std::ios::app);
    if (!results) {
        std::cerr << "Failed to open RESULTS.txt!\n";
        return 1;
    }
    // Redirect std::cout to the results file
    auto old_buf = std::cout.rdbuf(results.rdbuf());

    int N = 10'000'000;
    // Allocate and initialize host arrays
    float* h_a     = new float[N]; random_init(h_a, N);
    float* h_b     = new float[N]; random_init(h_b, N);
    float* h_c_gpu = new float[N];
    float* h_c_cpu = new float[N];
    float alpha    = 2.0f;

    // Allocate device arrays
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));
    hipMalloc(&d_c, N * sizeof(float));

    // Copy inputs to device
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch configuration: 256 threads per block
    dim3 blockDim(256);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x);

    // Time the GPU kernel
    auto t0 = high_resolution_clock::now();
    axpyKernel<<<gridDim, blockDim>>>(d_a, d_b, d_c, alpha, N);
    hipDeviceSynchronize();
    auto t1 = high_resolution_clock::now();
    long gpu_us = duration_cast<microseconds>(t1 - t0).count();

    // Copy result back
    hipMemcpy(h_c_gpu, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

    // Time the CPU version
    t0 = high_resolution_clock::now();
    axpy_cpu(h_a, h_b, h_c_cpu, alpha, N);
    t1 = high_resolution_clock::now();
    long cpu_us = duration_cast<microseconds>(t1 - t0).count();

    // Verify correctness
    float max_error = 0.0f;
    for (int i = 0; i < N; ++i) {
        max_error = std::max(max_error, std::fabs(h_c_gpu[i] - h_c_cpu[i]));
    }

    // Output benchmark and error
    std::cout << "AXPY on " << N << " elements (alpha=" << alpha << ")\n";
    std::cout << "\tGrid size (blocks)     = " << gridDim.x << "\n";
    std::cout << "\tBlock size (threads)   = " << blockDim.x << "\n";
    std::cout << "\tGPU time               = " << gpu_us << " µs\n";
    std::cout << "\tCPU time               = " << cpu_us << " µs\n";
    std::cout << "\tMaximum error          = " << max_error << "\n\n";

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c_gpu;
    delete[] h_c_cpu;

    // Restore std::cout
    std::cout.rdbuf(old_buf);
    return 0;
}
