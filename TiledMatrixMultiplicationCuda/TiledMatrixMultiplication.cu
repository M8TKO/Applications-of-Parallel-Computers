#include <iostream>
#include <random>
#include <cmath>    // for std::fabs
#include <hip/hip_runtime.h>

#define TILE_WIDTH 23

/**
 * CUDA kernel for tiled matrix multiplication:
 *   C = A × B
 * Matrices are square (N × N), stored row-major.
 * Handles cases where N is not a multiple of TILE_WIDTH by
 * padding out‐of‐bounds loads with zero.
 */
__global__
void matMulKernel(const float* A, const float* B, float* C, int N) {
    // Thread and block indices
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x,  by = blockIdx.y;
    // Global row and column this thread computes
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    // Shared tile buffers
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    float sum = 0.0f;
    int numTiles = (N + TILE_WIDTH - 1) / TILE_WIDTH;

    // Loop over all tile phases
    for (int t = 0; t < numTiles; ++t) {
        int aCol = t * TILE_WIDTH + tx;
        int bRow = t * TILE_WIDTH + ty;

        // Load A[row, aCol] into shared memory, or 0 if out of bounds
        if (row < N && aCol < N) {
            tileA[ty][tx] = A[row * N + aCol];
        } else {
            tileA[ty][tx] = 0.0f;
        }
        // Load B[bRow, col] into shared memory, or 0 if out of bounds
        if (bRow < N && col < N) {
            tileB[ty][tx] = B[bRow * N + col];
        } else {
            tileB[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Multiply the two tiles
        int effectiveWidth = min(TILE_WIDTH, N - t * TILE_WIDTH);
        if (row < N && col < N) {
            for (int k = 0; k < effectiveWidth; ++k) {
                sum += tileA[ty][k] * tileB[k][tx];
            }
        }

        __syncthreads();
    }

    // Write result if within bounds
    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

/**
 * Check CUDA API errors and exit on failure.
 */
inline void checkCuda(hipError_t status, const char* file, int line) {
    if (status != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(status)
                  << " at " << file << ":" << line << "\n";
        std::exit(EXIT_FAILURE);
    }
}
#define CUDA_CHECK(call) checkCuda((call), __FILE__, __LINE__)

/**
 * Host‐side reference matrix multiplication and result checker.
 * Returns true if max absolute error ≤ EPS × max value.
 */
bool checkResult(const float* A, const float* B, const float* C, int N) {
    const float EPS = 1e-6f;
    float maxError = 0.0f, maxValue = 0.0f;

    // Compute reference C_ref = A × B
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float tmp = 0.0f;
            for (int k = 0; k < N; ++k) {
                tmp += A[i * N + k] * B[k * N + j];
            }
            float diff = std::fabs(tmp - C[i * N + j]);
            float val  = std::fabs(tmp);
            maxError = std::max(maxError, diff);
            maxValue = std::max(maxValue, val);
        }
    }

    if (maxError > EPS * maxValue) {
        std::cout << "(Max error = " << maxError
                  << ", max value = " << maxValue
                  << ", rel. error = " << (maxError / maxValue) << ")\n";
        return false;
    }
    return true;
}

int main() {
    // Matrix dimension
    const int N = 1024;

    // Host allocations
    float* h_A = new float[N * N];
    float* h_B = new float[N * N];
    float* h_C = new float[N * N];

    // Random normal distribution generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dist(0.0f, 1.0f);

    // Initialize host matrices A and B
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = dist(gen);
        h_B[i] = dist(gen);
    }

    // Device allocations
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_B, N * N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_C, N * N * sizeof(float)));

    // Copy inputs to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice));

    // Configure kernel launch
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((N + TILE_WIDTH - 1) / TILE_WIDTH,
                 (N + TILE_WIDTH - 1) / TILE_WIDTH);

    // Create and record CUDA events for timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    // Launch the tiled matrix multiplication kernel
    matMulKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    // Measure elapsed time in milliseconds
    float milliseconds = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "GPU time: " << milliseconds << " ms\n";

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost));

    // Validate result on host
    bool ok = checkResult(h_A, h_B, h_C, N);
    std::cout << "Result is " << (ok ? "correct.\n" : "incorrect!\n");

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
